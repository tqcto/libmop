#include "hip/hip_runtime.h"
#include "../include/effect.cuh"

#include <stdio.h>

#include <hip/hip_runtime.h>
#include <>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define N 1 << 20

using namespace mop;

namespace mop_cuda {

	__host__ __device__ __forceinline__ int divUp(int total, int grain) {
		return (total + grain - 1) / grain;
	}

	__device__ void random(int* data, int max, int seed) {

		hiprandState_t state;
		hiprand_init(seed, 0, 0, &state);
		*data = hiprand(&state) % max;

	}
	__device__ void random(double* data, double max, int seed) {

		hiprandState_t state;
		hiprand_init(seed, 0, 0, &state);
		*data = (double)(hiprand(&state) % (int)(max * 10)) / 10.0;

	}

	__global__ void BlurDevice(uchar* src, uchar* dst, int w, int h, int c, int m, int n) {

		int x = blockIdx.x * blockDim.x + threadIdx.x;
		int y = blockIdx.y * blockDim.y + threadIdx.y;

		int data = 0,
			num = 1;

		for (int cc = 0; cc < c; cc++) {

			data = 0;
			num = 1;

			for (int yy = y - m; yy <= y + n; yy++) {
				if (0 <= yy && yy < h) {
					for (int xx = x - m; xx <= x + n; xx++) {
						if (0 <= xx && xx < w) {
							data += src[(xx + yy * w) * c + cc];
							num++;
						}
					}
				}
			}

			dst[(x + y * w) * c + cc] = data / num;

		}

	}

	DLL_EXPORT void Blur(matrix* src, matrix* dst, int amount) {

		int w = src->width(),
			h = src->height(),
			c = src->channel();

		if (w && h && c && amount) {

			int m = 0, n = 0;
			if ((amount * 5) % 10 != 0) {
				m = amount / 2;
				n = m;
			}
			else {
				m = (double)amount / 2.0;
				n = m - 1;
			}

			uchar* input, * output;
			int size = sizeof(uchar) * w * h * c;

			hipMalloc((void**)&input, size);
			hipMalloc((void**)&output, size);

			hipMemcpy(input, src->data, size, hipMemcpyHostToDevice);

			*dst = matrix(w, h, c);

			///*
			const dim3 block(w / 125, h / 125);
			const dim3 grid(divUp(w, block.x), divUp(h, block.y));
			//*/
			/*
			int nw = w * 2048;
			int nh = h * 2048;
			const dim3 block(w, h);
			const dim3 grid(nw / block.x, nh / block.y);
			*/
			/*
			const dim3 block(1, 1);
			const dim3 grid(w, h);
			*/

			BlurDevice << <grid, block >> > (input, output, w, h, c, m, n);

			hipMemcpy(
				dst->data,
				output,
				size,
				hipMemcpyDeviceToHost
			);

			hipFree(input);
			hipFree(output);

		}

	}
	DLL_EXPORT void Blur(uchar** src, uchar** dst, int w, int h, int c, int amount) {

		if (w && h && c && amount) {

			int m = 0, n = 0;
			if ((amount * 5) % 10 != 0) {
				m = amount / 2;
				n = m;
			}
			else {
				m = (double)amount / 2.0;
				n = m - 1;
			}

			uchar* input, * output;
			int size = sizeof(uchar) * w * h * c;

			hipMalloc((void**)&input, size);
			hipMalloc((void**)&output, size);

			hipMemcpy(input, *src, size, hipMemcpyHostToDevice);

			const dim3 block(w / 125, h / 125);
			const dim3 grid(divUp(w, block.x), divUp(h, block.y));

			BlurDevice << <grid, block >> > (input, output, w, h, c, m, n);

			hipMemcpy(
				*dst,
				output,
				size,
				hipMemcpyDeviceToHost
			);

			hipFree(input);
			hipFree(output);

		}

	}

	__global__ void Test_gpu(int* src, int* dst, int* grid, int* block, int* thread) {

		/*
		* blockIdx	= block index
		* blockDim	= *grid index* or *block size*
		* threadIdx	= thread index
		*/
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		dst[index] = src[4 - index];

		grid[index] = blockDim.x;
		block[index] = blockIdx.x;
		thread[index] = threadIdx.x;

	}

	DLL_EXPORT void Test() {

		int source[5] = { 0, 1, 2, 3, 4 };
		int grid[5], block[5], thread[5];

		int *src, *dst, *ggrid, * gblock, *gthread;

		hipMalloc((void**)&src, sizeof(int) * 5);
		hipMalloc((void**)&dst, sizeof(int) * 5);

		hipMalloc((void**)&ggrid,		sizeof(int) * 5);
		hipMalloc((void**)&gblock,		sizeof(int) * 5);
		hipMalloc((void**)&gthread,	sizeof(int) * 5);

		const dim3 _block(5);
		const dim3 _grid(N / _block.x);

		hipMemcpy(src, source, sizeof(int) * 5, hipMemcpyHostToDevice);

		Test_gpu << <_grid, _block >> > (src, dst, ggrid, gblock, gthread);

		hipMemcpy(source, dst, sizeof(int) * 5, hipMemcpyDeviceToHost);

		hipMemcpy(block,	gblock,		sizeof(int) * 5, hipMemcpyDeviceToHost);
		hipMemcpy(grid,	ggrid,		sizeof(int) * 5, hipMemcpyDeviceToHost);
		hipMemcpy(thread,	gthread,	sizeof(int) * 5, hipMemcpyDeviceToHost);

		hipFree(src);
		hipFree(dst);

		hipFree(ggrid);
		hipFree(gblock);
		hipFree(gthread);

		printf("[source]\n");
		for (int i = 0; i < 5; i++) {
			printf("%d ", source[i]);
		}
		printf("\n");

		printf("\n[grid]");
		for (int i = 0; i < 5; i++) {
			printf(" %d", grid[i]);
		}
		printf("\n[block]");
		for (int i = 0; i < 5; i++) {
			printf(" %d", block[i]);
		}
		printf("\n[thread]");
		for (int i = 0; i < 5; i++) {
			printf(" %d", thread[i]);
		}
		printf("\n");

	}

}
